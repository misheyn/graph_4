#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <GL/freeglut.h>

#define PART_COUNT 1000
clock_t startTime;

typedef struct particle {
    double posX;
    double posX0;
    double posY;
    double posY0;
    double vx;
    double q;
    double m;
} particle;

particle *part = nullptr;
particle *partGPU = nullptr;
int *sizeGPU = nullptr;
clock_t *timeGPU = nullptr;

void moveParticle(particle *arr, int size, clock_t time) {
    for (int i = 0; i < size; ++i) {
        if (arr[i].posX > -0.5)
            arr[i].posY = arr[i].posY0 +
                          (arr[i].q * 0.001 * ((double) time / 10000.0) * ((double) time / 10000.0)) / (2 * arr[i].m);
        arr[i].posX = arr[i].posX0 + arr[i].vx * (double) time / 10000.0;
    }
}

__global__ void moveParticleGPU(particle *arr, int *size, clock_t *time) {
    unsigned th = threadIdx.x;
    unsigned bl = blockIdx.x;
    unsigned i = bl * 1024 + th;
    if (i < *size) {
        if (arr[i].posX > -0.5)
            arr[i].posY = arr[i].posY0 +
                          (arr[i].q * 0.001 * ((double) *time / 10000.0) * ((double) *time / 10000.0)) / (2 * arr[i].m);
        arr[i].posX = arr[i].posX0 + arr[i].vx * (double) *time / 10000.0;
    }
}

void initParticles() {
    part = (particle *) malloc(sizeof(particle) * PART_COUNT);
    hipMalloc((void **)&partGPU, sizeof(particle) * PART_COUNT);
    hipMalloc((void **)&sizeGPU, sizeof(int));
    hipMalloc((void **)&timeGPU, sizeof(clock_t));

    for (int i = 0; i < PART_COUNT; i++) {
        part[i].posY0 = (-30 + rand() % 60) / 100.0;
        part[i].posX0 = -1;
        part[i].vx = (10 + rand() % 200) / 2000.0;
        part[i].posX = part[i].posX0;
        part[i].posY = part[i].posY0;
        part[i].m = (rand() % 1000) / 10000000000.0;
        part[i].q = (-1000 + rand() % 2000) / 10000000000.0;
    }
}

void display() {
    int size = PART_COUNT;
    clock_t time = clock() - startTime;
    unsigned threadX, blockX;

    glPushMatrix();
    glEnable(GL_POINT_SMOOTH);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (size > 1024) {
        threadX = 1024;
        blockX = 1 + (size) / 1024;
    } else {
        threadX = size;
        blockX = 1;
    }
    hipMemcpy(partGPU, part, sizeof (particle) * PART_COUNT, hipMemcpyHostToDevice);
    hipMemcpy(sizeGPU, &size, sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(timeGPU, &time, sizeof (clock_t), hipMemcpyHostToDevice);

    moveParticleGPU<<<blockX, threadX>>>(partGPU, sizeGPU, timeGPU);

    hipDeviceSynchronize();
    hipMemcpy(part, partGPU, sizeof (particle) * PART_COUNT, hipMemcpyDeviceToHost);

    glBegin(GL_LINES);
    glColor3d(1, 1, 1);
    glVertex2d(-0.5, -1);
    glVertex2d(-0.5, -0.3);
    glVertex2d(-0.5, 0.3);
    glVertex2d(-0.5, 1);
    glEnd();

    glBegin(GL_POINTS);
    for (int i = 0; i < PART_COUNT; ++i) {
        if (part[i].q < 0)
            glColor3d(0, 0, 1);
        else
            glColor3d(0, 1, 0);
        glVertex2d(part[i].posX, part[i].posY);
    }
    glEnd();

    glPopMatrix();
    glutSwapBuffers();
}

void timer() {
    glutPostRedisplay();
    glutTimerFunc(10, reinterpret_cast<void (*)(int)>(timer), 0);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    initParticles();
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(900, 700);
    glutInitWindowPosition(450, 60);
    glutCreateWindow("Particles in an electric field");
    glClearColor(0, 0, 0, 0);
    glutDisplayFunc(display);
    glutTimerFunc(10, reinterpret_cast<void (*)(int)>(timer), 0);
    startTime = clock();
    glutMainLoop();
}
